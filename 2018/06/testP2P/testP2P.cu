
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv)
{
    printf("[%s] - Starting...\n", argv[0]);
    int gpuid[2] = {1, 2}; // we want to find the first two GPU's that can support P2P
    float total_time = 0.0;
    bool enable_p2p = true;

    if (enable_p2p)
    {
        // Enable peer access
        printf("Enabling peer access between GPU%d and GPU%d...\n", gpuid[0], gpuid[1]);
        checkCudaErrors(hipSetDevice(gpuid[0]));
        checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[1], 0));
        checkCudaErrors(hipSetDevice(gpuid[1]));
        checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[0], 0));
    }

    for (int loop = 0; loop < 100; loop++)
    {
        // Allocate buffers
        const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
        printf("Allocating buffers (%iMB on GPU%d, GPU%d and CPU Host)...\n", int(buf_size / 1024 / 1024), gpuid[0], gpuid[1]);
        checkCudaErrors(hipSetDevice(gpuid[0]));
        float *g0;
        checkCudaErrors(hipMalloc(&g0, buf_size));
        checkCudaErrors(hipSetDevice(gpuid[1]));
        float *g1;
        checkCudaErrors(hipMalloc(&g1, buf_size));

        // Create CUDA event handles
        hipEvent_t start_event, stop_event;
        float time_memcpy;
        int eventflags = hipEventBlockingSync;
        checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
        checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

        // P2P memcopy() benchmark
        checkCudaErrors(hipEventRecord(start_event, 0));

        for (int i = 0; i < 100; i++)
        {
            // With UVA we don't need to specify source and target devices, the
            // runtime figures this out by itself from the pointers

            // Ping-pong copy between GPUs
            if (i % 2 == 0)
            {
                checkCudaErrors(hipMemcpy(g1, g0, buf_size, hipMemcpyDefault));
            }
            else
            {
                checkCudaErrors(hipMemcpy(g0, g1, buf_size, hipMemcpyDefault));
            }
        }

        checkCudaErrors(hipEventRecord(stop_event, 0));
        checkCudaErrors(hipEventSynchronize(stop_event));
        checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
        total_time += time_memcpy;
        printf("hipMemcpyPeer / hipMemcpy (%f ms) between GPU%d and GPU%d: %.2fGB/s\n", time_memcpy, gpuid[0], gpuid[1],
               (1.0f / (time_memcpy / 1000.0f)) * ((100.0f * buf_size)) / 1024.0f / 1024.0f / 1024.0f);

        // Free resources
        checkCudaErrors(hipEventDestroy(start_event));
        checkCudaErrors(hipEventDestroy(stop_event));
        checkCudaErrors(hipSetDevice(gpuid[0]));
        checkCudaErrors(hipFree(g0));
        checkCudaErrors(hipSetDevice(gpuid[1]));
        checkCudaErrors(hipFree(g1));
    }


    if (enable_p2p)
    {
        // Disable peer access (also unregisters memory for non-UVA cases)
        printf("Disabling peer access...\n");
        checkCudaErrors(hipSetDevice(gpuid[0]));
        checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[1]));
        checkCudaErrors(hipSetDevice(gpuid[1]));
        checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[0]));
    }

    printf("Total time is %.2fs\n", total_time / 1000);
    //delete device_handler;
    return (EXIT_SUCCESS);
}
